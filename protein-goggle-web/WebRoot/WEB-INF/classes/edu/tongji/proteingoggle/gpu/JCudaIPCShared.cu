
#include <hip/hip_runtime.h>
extern "C"
__global__ void ipc(double *CMass, double *SMass, double *HMass, double *NMass, double *OMass, double *CP, double *SP, double *HP, double *NP, double *OP, int *arrayCombination, int *indexCombination, double *MassOutput, double *POutput)
{

	int cMax = arrayCombination[0];
	int sMax = arrayCombination[1];
	int hMax = arrayCombination[2];
	int nMax = arrayCombination[3];
	int oMax = arrayCombination[4];
	
	double *LocalCMass = new double[cMax];
	double *LocalSMass = new double[sMax];
	double *LocalHMass = new double[hMax];
	double *LocalNMass = new double[nMax];
	double *LocalOMass = new double[oMax];
	
	double *LocalCP = new double[cMax];
	double *LocalSP = new double[sMax];
	double *LocalHP = new double[hMax];
	double *LocalNP = new double[nMax];
	double *LocalOP = new double[oMax];
	
	int index = blockIdx.x;
	int indexC = index * 5;
	int indexS = index * 5 + 1;
	int indexH = index * 5 + 2;
	int indexN = index * 5 + 3;
	int indexO = index * 5 + 4;
	
	int CNumber = indexCombination[indexC];
	int SNumber = indexCombination[indexS];
	int HNumber = indexCombination[indexH];
	int NNumber = indexCombination[indexN];
	int ONumber = indexCombination[indexO];
	
	for(int i = 0; i != cMax; i++){
		LocalCMass[i] = CMass[cMax * (CNumber - 1) + i];
		LocalCP[i] = CP[cMax * (CNumber - 1) + i];
	}
	for(int i = 0; i != sMax; i++){
		if(SNumber > 0){
			LocalSMass[i] = SMass[sMax * (SNumber - 1) + i];
			LocalSP[i] = SP[sMax * (SNumber - 1) + i];
		}else{
			LocalSMass[i] = 0;
			LocalSP[i] = 1;
		}
	}
	for(int i = 0; i != hMax; i++){
		LocalHMass[i] = HMass[hMax * (HNumber - 1) + i];
		LocalHP[i] = HP[hMax * (HNumber - 1) + i];
	}
	for(int i = 0; i != nMax; i++){
		LocalNMass[i] = NMass[nMax * (NNumber - 1) + i];
		LocalNP[i] = NP[nMax * (NNumber - 1) + i];
	}
	for(int i = 0; i != oMax; i++){
		LocalOMass[i] = OMass[oMax * (ONumber - 1) + i];
		LocalOP[i] = OP[oMax * (ONumber - 1) + i];
	}
	
	int count = 0;
	int total = cMax * sMax * hMax * nMax * oMax;
	for(int i = 0; i != cMax; i++){
		for(int j = 0; j != sMax; j++){
			for(int k = 0; k != hMax; k++){
				for(int l = 0; l != nMax; l++){
					for(int m = 0; m != oMax; m++){
						if(SNumber > 0){
							MassOutput[total * index + count] = LocalCMass[i] + LocalSMass[j] + LocalHMass[k] + LocalNMass[l] + LocalOMass[m] + 5 * 0.00054858;
						}else{
							MassOutput[total * index + count] = LocalCMass[i] + LocalSMass[j] + LocalHMass[k] + LocalNMass[l] + LocalOMass[m] + 4 * 0.00054858;
						}
						
						POutput[total * index + count] = LocalCP[i] * LocalSP[j] * LocalHP[k] * LocalNP[l] * LocalOP[m];
						count++;
					}
				}
			}
		}
	}
}
